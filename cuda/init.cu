#include <assert.h>
#include "kem.h"

int pqcrystals_init()
{
    assert( !hipFree(0));

    assert( !hipFuncSetCacheConfig(reinterpret_cast<const void*>( crypto_kem_keypair_kernel), hipFuncCachePreferL1));
    assert( !hipFuncSetCacheConfig(reinterpret_cast<const void*>( crypto_kem_enc_kernel), hipFuncCachePreferL1));
    assert( !hipFuncSetCacheConfig(reinterpret_cast<const void*>( crypto_kem_dec_kernel), hipFuncCachePreferL1));

    return 0;
}
