#include "hip/hip_runtime.h"
#include <stddef.h>
#include <stdint.h>
#include <string.h>
#include <assert.h>
#include <stdio.h>
#include "params.h"
#include "kem.h"
#include "indcpa.h"
#include "verify.h"
#include "symmetric.h"
#include "randombytes.h"

#include <hip/hip_runtime.h>

#define GPU_ASSERT(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*************************************************
* Name:        crypto_kem_keypair_derand
*
* Description: Generates public and private key
*              for CCA-secure Kyber key encapsulation mechanism
*
* Arguments:   - uint8_t *pk: pointer to output public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes per thread)
*              - uint8_t *sk: pointer to output private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes per thread)
*              - uint8_t *coins: pointer to input randomness
*                (an already allocated array filled with 2*KYBER_SYMBYTES random bytes per thread)
**
* Returns 0 (success)
**************************************************/
__global__ void
crypto_kem_keypair_derand(uint8_t *pk,
                          uint8_t *sk,
                          const uint8_t *coins,
                          uint32_t keypair_count)
{
  const int tid = threadIdx.x;
  const int block_size = blockDim.x;
  const int bid = blockIdx.x;
  const int coins_offset = ( bid * block_size + tid ) * 2 * KYBER_SYMBYTES;
  const int sk_offset = ( bid * block_size + tid ) * KYBER_SECRETKEYBYTES;
  const int pk_offset = ( bid * block_size + tid ) * KYBER_PUBLICKEYBYTES;

  if ( bid * block_size + tid < keypair_count )
  {
    indcpa_keypair_derand(pk + pk_offset, sk + sk_offset, coins + coins_offset);
    memcpy(sk + sk_offset + KYBER_INDCPA_SECRETKEYBYTES, pk + pk_offset, KYBER_PUBLICKEYBYTES);
    hash_h(sk + sk_offset + KYBER_SECRETKEYBYTES - 2 * KYBER_SYMBYTES, pk + pk_offset, KYBER_PUBLICKEYBYTES);
    /* Value z for pseudo-random output on reject */
    memcpy(sk + sk_offset + KYBER_SECRETKEYBYTES - KYBER_SYMBYTES, coins + coins_offset + KYBER_SYMBYTES, KYBER_SYMBYTES);
  }
}

#define MAX_BLOCK_SIZE 128

/*************************************************
* Name:        crypto_kem_keypair
*
* Description: Generates public and private key
*              for CCA-secure Kyber key encapsulation mechanism
*
* Arguments:   - uint8_t *pk: pointer to output public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*              - uint8_t *sk: pointer to output private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes)
*
* Returns 0 (success)
**************************************************/
int crypto_kem_keypair(uint8_t *pk,
                       uint8_t *sk,
                       const uint8_t *coins,
                       uint32_t keypair_count)
{
  dim3 block_dim (1, 1, 1);
  dim3 grid_dim (1, 1, 1);

  if (keypair_count <= MAX_BLOCK_SIZE) {
    block_dim.x = keypair_count;
    grid_dim.x = 1;
  } else {
    block_dim.x = MAX_BLOCK_SIZE;
    grid_dim.x = ( keypair_count + MAX_BLOCK_SIZE - 1 ) / MAX_BLOCK_SIZE;
  }

  uint8_t *d_pk = nullptr;
  uint8_t *d_sk = nullptr;
  uint8_t *d_coins = nullptr;

  hipMalloc( &d_pk, keypair_count * CRYPTO_PUBLICKEYBYTES);
  hipMalloc( &d_sk, keypair_count * CRYPTO_SECRETKEYBYTES);
  hipMalloc( &d_coins, keypair_count * 2 * KYBER_SYMBYTES);
  assert( d_pk && d_sk && d_coins);

  GPU_ASSERT( hipMemcpy( d_coins, coins, keypair_count * 2 * KYBER_SYMBYTES, hipMemcpyHostToDevice) );

  printf( "gridDim (%d %d %d), blockDim (%d %d %d)\n", grid_dim.x, grid_dim.y, grid_dim.z, block_dim.x, block_dim.y, block_dim.z);
  crypto_kem_keypair_derand<<<grid_dim, block_dim>>>( d_pk, d_sk, d_coins, keypair_count);

  GPU_ASSERT( hipGetLastError() );

  GPU_ASSERT( hipMemcpy( pk, d_pk, keypair_count * CRYPTO_PUBLICKEYBYTES, hipMemcpyDeviceToHost) );
  GPU_ASSERT( hipMemcpy( sk, d_sk, keypair_count * CRYPTO_SECRETKEYBYTES, hipMemcpyDeviceToHost) );
  hipFree( d_pk);
  hipFree( d_sk);
  hipFree( d_coins);
  return 0;
}

#if 0
/*************************************************
* Name:        crypto_kem_enc_derand
*
* Description: Generates cipher text and shared
*              secret for given public key
*
* Arguments:   - uint8_t *ct: pointer to output cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *pk: pointer to input public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*              - const uint8_t *coins: pointer to input randomness
*                (an already allocated array filled with KYBER_SYMBYTES random bytes)
**
* Returns 0 (success)
**************************************************/
__device__ void
crypto_kem_enc_derand(uint8_t *ct,
                      uint8_t *ss,
                      const uint8_t *pk,
                      const uint8_t *coins,
                      uint32_t keypair_count)
{
  const int tid = threadIdx.x;
  const int block_size = blockDim.x;
  const int bid = blockIdx.x;
  const int coins_offset = ( bid * block_size + tid ) * 2 * KYBER_SYMBYTES;
  const int sk_offset = ( bid * block_size + tid ) * KYBER_SECRETKEYBYTES;
  const int pk_offset = ( bid * block_size + tid ) * KYBER_PUBLICKEYBYTES;

  if ( bid * block_size + tid < keypair_count )
  {
      uint8_t buf[2*KYBER_SYMBYTES];
      /* Will contain key, coins */
      uint8_t kr[2*KYBER_SYMBYTES];

      memcpy(buf, coins, KYBER_SYMBYTES);

      /* Multitarget countermeasure for coins + contributory KEM */
      hash_h(buf+KYBER_SYMBYTES, pk, KYBER_PUBLICKEYBYTES);
      hash_g(kr, buf, 2*KYBER_SYMBYTES);

      /* coins are in kr+KYBER_SYMBYTES */
      indcpa_enc(ct, buf, pk, kr+KYBER_SYMBYTES);

      memcpy(ss,kr,KYBER_SYMBYTES);
  }
}

/*************************************************
* Name:        crypto_kem_enc
*
* Description: Generates cipher text and shared
*              secret for given public key
*
* Arguments:   - uint8_t *ct: pointer to output cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *pk: pointer to input public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*
* Returns 0 (success)
**************************************************/
int crypto_kem_enc(uint8_t *ct,
                   uint8_t *ss,
                   const uint8_t *pk)
{
  uint8_t coins[KYBER_SYMBYTES];
  randombytes(coins, KYBER_SYMBYTES);
  crypto_kem_enc_derand(ct, ss, pk, coins);
  return 0;
}

/*************************************************
* Name:        crypto_kem_dec
*
* Description: Generates shared secret for given
*              cipher text and private key
*
* Arguments:   - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *ct: pointer to input cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - const uint8_t *sk: pointer to input private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes)
*
* Returns 0.
*
* On failure, ss will contain a pseudo-random value.
**************************************************/
int crypto_kem_dec(uint8_t *ss,
                   const uint8_t *ct,
                   const uint8_t *sk)
{
  int fail;
  uint8_t buf[2*KYBER_SYMBYTES];
  /* Will contain key, coins */
  uint8_t kr[2*KYBER_SYMBYTES];
  uint8_t cmp[KYBER_CIPHERTEXTBYTES+KYBER_SYMBYTES];
  const uint8_t *pk = sk+KYBER_INDCPA_SECRETKEYBYTES;

  indcpa_dec(buf, ct, sk);

  /* Multitarget countermeasure for coins + contributory KEM */
  memcpy(buf+KYBER_SYMBYTES, sk+KYBER_SECRETKEYBYTES-2*KYBER_SYMBYTES, KYBER_SYMBYTES);
  hash_g(kr, buf, 2*KYBER_SYMBYTES);

  /* coins are in kr+KYBER_SYMBYTES */
  indcpa_enc(cmp, buf, pk, kr+KYBER_SYMBYTES);

  fail = verify(ct, cmp, KYBER_CIPHERTEXTBYTES);

  /* Compute rejection key */
  rkprf(ss,sk+KYBER_SECRETKEYBYTES-KYBER_SYMBYTES,ct);

  /* Copy true key to return buffer if fail is false */
  cmov(ss,kr,KYBER_SYMBYTES,!fail);

  return 0;
}
#endif
