#include "hip/hip_runtime.h"
#include <stddef.h>
#include <stdint.h>
#include <string.h>
#include <assert.h>
#include <stdio.h>
#include <vector>
#include "params.h"
#include "kem.h"
#include "indcpa.h"
#include "verify.h"
#include "symmetric.h"
#include "randombytes.h"

#include <hip/hip_runtime.h>

#define GPU_ASSERT(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*************************************************
* Name:        crypto_kem_keypair_kernel
*
* Description: Generates public and private key
*              for CCA-secure Kyber key encapsulation mechanism
*
* Arguments:   - uint8_t *pk: pointer to output public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes per thread)
*              - uint8_t *sk: pointer to output private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes per thread)
*              - uint8_t *coins: pointer to input randomness
*                (an already allocated array filled with 2*KYBER_SYMBYTES random bytes per thread)
*              - uint32_t n_keypair: amount of keys to generate
*
* Returns 0 (success)
**************************************************/
__global__ void
crypto_kem_keypair_kernel(uint8_t *pk,
                          uint8_t *sk,
                          const uint8_t *coins,
                          uint32_t n_keypair)
{
  const int tid = threadIdx.x;
  const int block_size = blockDim.x;
  const int bid = blockIdx.x;

  const int idx = bid * block_size + tid;
  uint8_t       *thread_pk    = pk    + idx * KYBER_PUBLICKEYBYTES;
  uint8_t       *thread_sk    = sk    + idx * KYBER_SECRETKEYBYTES;
  const uint8_t *thread_coins = coins + idx * 2*KYBER_SYMBYTES;

  if ( idx < n_keypair )
  {
    indcpa_keypair_derand(thread_pk, thread_sk, thread_coins);
    memcpy(thread_sk + KYBER_INDCPA_SECRETKEYBYTES, thread_pk, KYBER_PUBLICKEYBYTES);
    hash_h(thread_sk + KYBER_SECRETKEYBYTES - 2*KYBER_SYMBYTES, thread_pk, KYBER_PUBLICKEYBYTES);
    /* Value z for pseudo-random output on reject */
    memcpy(thread_sk + KYBER_SECRETKEYBYTES - KYBER_SYMBYTES, thread_coins + KYBER_SYMBYTES, KYBER_SYMBYTES);
  }
}

#define MAX_BLOCK_SIZE 128
const uint32_t kThreadsPerBlock = 32;
const uint32_t kBlocksPerGrid = 256;
const uint32_t kKeysPerGrid = kBlocksPerGrid * kThreadsPerBlock;

/*************************************************
* Name:        crypto_kem_keypair
*
* Description: Generates public and private key
*              for CCA-secure Kyber key encapsulation mechanism
*
* Arguments:   - uint8_t *pk: pointer to output public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*              - uint8_t *sk: pointer to output private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes)
*
* Returns 0 (success)
**************************************************/
int crypto_kem_keypair(uint8_t *pk,
                       uint8_t *sk,
                       const uint8_t *coins,
                       uint32_t n_keypair)
{
  const uint32_t n_streams = (n_keypair + kKeysPerGrid - 1) / kKeysPerGrid;
  const uint32_t tail_keypairs = n_keypair % kKeysPerGrid;

  uint8_t *d_pk = nullptr;
  uint8_t *d_sk = nullptr;
  uint8_t *d_coins = nullptr;

  hipMalloc( &d_pk, n_keypair * KYBER_PUBLICKEYBYTES);
  hipMalloc( &d_sk, n_keypair * KYBER_SECRETKEYBYTES);
  hipMalloc( &d_coins, n_keypair * 2 * KYBER_SYMBYTES);
  assert( d_pk && d_sk && d_coins);

  GPU_ASSERT( hipMemcpy( d_coins, coins, n_keypair * 2 * KYBER_SYMBYTES, hipMemcpyHostToDevice) );

  std::vector<hipStream_t> streams( n_streams);
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    GPU_ASSERT( hipStreamCreate( &streams[i]));
  }

  auto getNKeysForGrid = [=]( uint32_t i) -> uint32_t
  {
    if ( (tail_keypairs != 0) && (i == n_streams - 1) )
    {
      return tail_keypairs;
    } else {
      return kKeysPerGrid;
    }
  };

  // Launch all grids in streams.
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    uint32_t offset = i * kKeysPerGrid;
    uint32_t n_cur = getNKeysForGrid( i);

    uint8_t *d_grid_pk    = d_pk    + offset * KYBER_PUBLICKEYBYTES;
    uint8_t *d_grid_sk    = d_sk    + offset * KYBER_SECRETKEYBYTES;
    uint8_t *d_grid_coins = d_coins + offset * 2*KYBER_SYMBYTES;

    printf( "keygen gridDim (%u), blockDim (%u), stream launch (%u), #keypairs on grid (%u keys), offset (%u keys)\n",
            kBlocksPerGrid, kThreadsPerBlock, i, n_cur, offset);
    crypto_kem_keypair_kernel<<<kBlocksPerGrid, kThreadsPerBlock, 0, streams[i]>>>(
        d_grid_pk,
        d_grid_sk,
        d_grid_coins,
        n_cur
    );
  }

  // Launch copy for result of every stream.
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    uint32_t offset = i * kKeysPerGrid;
    uint32_t n_cur = getNKeysForGrid( i);

    uint8_t *h_grid_pk = pk   + offset * KYBER_PUBLICKEYBYTES;
    uint8_t *h_grid_sk = sk   + offset * KYBER_SECRETKEYBYTES;
    uint8_t *d_grid_pk = d_pk + offset * KYBER_PUBLICKEYBYTES;
    uint8_t *d_grid_sk = d_sk + offset * KYBER_SECRETKEYBYTES;

    GPU_ASSERT( hipMemcpyAsync( h_grid_pk,
                                 d_grid_pk,
                                 n_cur * KYBER_PUBLICKEYBYTES,
                                 hipMemcpyDeviceToHost,
                                 streams[i]));

    GPU_ASSERT( hipMemcpyAsync( h_grid_sk,
                                 d_grid_sk,
                                 n_cur * KYBER_SECRETKEYBYTES,
                                 hipMemcpyDeviceToHost,
                                 streams[i]));
  }

  // Wait for all streams.
  GPU_ASSERT( hipDeviceSynchronize());

  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    GPU_ASSERT( hipStreamDestroy( streams[i]));
  }

  hipFree( d_pk);
  hipFree( d_sk);
  hipFree( d_coins);

  return 0;
}

/*************************************************
* Name:        crypto_kem_enc_kernel
*
* Description: Generates cipher text and shared
*              secret for given public key
*
* Arguments:   - uint8_t *ct: pointer to output cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *pk: pointer to input public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*              - const uint8_t *coins: pointer to input randomness
*                (an already allocated array filled with KYBER_SYMBYTES random bytes)
**
* Returns 0 (success)
**************************************************/
__global__ void
crypto_kem_enc_kernel(uint8_t *ct,
                      uint8_t *ss,
                      const uint8_t *pk,
                      const uint8_t *coins,
                      uint32_t n_keypair)
{
  const int tid = threadIdx.x;
  const int block_size = blockDim.x;
  const int bid = blockIdx.x;
  const int coins_offset = ( bid * block_size + tid ) * KYBER_SYMBYTES;
  const int pk_offset = ( bid * block_size + tid ) * KYBER_PUBLICKEYBYTES;
  const int ct_offset = ( bid * block_size + tid ) * KYBER_INDCPA_BYTES;
  const int ss_offset = ( bid * block_size + tid ) * KYBER_SSBYTES;

  if ( bid * block_size + tid < n_keypair )
  {
      uint8_t buf[2*KYBER_SYMBYTES];
      /* Will contain key, coins */
      uint8_t kr[2*KYBER_SYMBYTES];

      memcpy(buf, coins + coins_offset, KYBER_SYMBYTES);

      /* Multitarget countermeasure for coins + contributory KEM */
      hash_h(buf + KYBER_SYMBYTES, pk + pk_offset, KYBER_PUBLICKEYBYTES);
      hash_g(kr, buf, 2*KYBER_SYMBYTES);

      /* coins are in kr+KYBER_SYMBYTES */
      indcpa_enc(ct + ct_offset, buf, pk + pk_offset, kr + KYBER_SYMBYTES);

      memcpy(ss + ss_offset,kr,KYBER_SYMBYTES);
  }
}

/*************************************************
* Name:        crypto_kem_enc
*
* Description: Generates cipher text and shared
*              secret for given public key
*
* Arguments:   - uint8_t *ct: pointer to output cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *pk: pointer to input public key
*                (an already allocated array of KYBER_PUBLICKEYBYTES bytes)
*
* Returns 0 (success)
**************************************************/
int crypto_kem_enc(uint8_t *ct,
                   uint8_t *ss,
                   const uint8_t *pk,
                   const uint8_t *coins,
                   uint32_t n_keypair)
{
  const uint32_t n_streams = (n_keypair + kKeysPerGrid - 1) / kKeysPerGrid;
  const uint32_t tail_keypairs = n_keypair % kKeysPerGrid;

  uint8_t *d_ct = nullptr;
  uint8_t *d_ss = nullptr;
  uint8_t *d_pk = nullptr;
  uint8_t *d_coins = nullptr;

  hipMalloc( &d_ct, n_keypair * KYBER_CIPHERTEXTBYTES);
  hipMalloc( &d_ss, n_keypair * KYBER_SSBYTES);
  hipMalloc( &d_pk, n_keypair * KYBER_PUBLICKEYBYTES);
  hipMalloc( &d_coins, n_keypair * KYBER_SYMBYTES);
  assert( d_ct && d_ss && d_pk && d_coins);

  GPU_ASSERT( hipMemcpy( d_pk, pk, n_keypair * KYBER_PUBLICKEYBYTES, hipMemcpyHostToDevice) );
  GPU_ASSERT( hipMemcpy( d_coins, coins, n_keypair * KYBER_SYMBYTES, hipMemcpyHostToDevice) );

  std::vector<hipStream_t> streams( n_streams);
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    GPU_ASSERT( hipStreamCreate( &streams[i]));
  }

  auto getNKeysForGrid = [=]( uint32_t i) -> uint32_t
  {
    if ( (tail_keypairs != 0) && (i == n_streams - 1) )
    {
      return tail_keypairs;
    } else {
      return kKeysPerGrid;
    }
  };

  // Launch all grids in streams.
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    uint32_t offset = i * kKeysPerGrid;
    uint32_t n_cur = getNKeysForGrid( i);

    uint8_t *d_grid_ct    = d_ct    + offset * KYBER_CIPHERTEXTBYTES;
    uint8_t *d_grid_ss    = d_ss    + offset * KYBER_SSBYTES;
    uint8_t *d_grid_pk    = d_pk    + offset * KYBER_PUBLICKEYBYTES;
    uint8_t *d_grid_coins = d_coins + offset * KYBER_SYMBYTES;

    printf( "enc gridDim (%u), blockDim (%u), stream launch (%u), #keys on grid (%u keys), offset (%u keys)\n",
            kBlocksPerGrid, kThreadsPerBlock, i, n_cur, offset);
    crypto_kem_enc_kernel<<<kBlocksPerGrid, kThreadsPerBlock, 0, streams[i]>>>(
        d_grid_ct,
        d_grid_ss,
        d_grid_pk,
        d_grid_coins,
        n_cur
    );
  }

  // Launch copy for result of every stream.
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    uint32_t offset = i * kKeysPerGrid;
    uint32_t n_cur = getNKeysForGrid( i);

    uint8_t *h_grid_ct = ct   + offset * KYBER_CIPHERTEXTBYTES;
    uint8_t *h_grid_ss = ss   + offset * KYBER_SSBYTES;
    uint8_t *d_grid_ct = d_ct + offset * KYBER_CIPHERTEXTBYTES;
    uint8_t *d_grid_ss = d_ss + offset * KYBER_SSBYTES;

    GPU_ASSERT( hipMemcpyAsync( h_grid_ct,
                                 d_grid_ct,
                                 n_cur * KYBER_CIPHERTEXTBYTES,
                                 hipMemcpyDeviceToHost,
                                 streams[i]));

    GPU_ASSERT( hipMemcpyAsync( h_grid_ss,
                                 d_grid_ss,
                                 n_cur * KYBER_SSBYTES,
                                 hipMemcpyDeviceToHost,
                                 streams[i]));
  }

  // Wait for all streams.
  GPU_ASSERT( hipDeviceSynchronize());

  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    GPU_ASSERT( hipStreamDestroy( streams[i]));
  }

  hipFree( d_ct);
  hipFree( d_ss);
  hipFree( d_pk);
  hipFree( d_coins);

  return 0;
}

/*************************************************
* Name:        crypto_kem_dec_kernel
*
* Description: Generates shared secret for given
*              cipher text and private key
*
* Arguments:   - uint8_t *ss: pointer to output shared secret
*                (an already allocated array of KYBER_SSBYTES bytes)
*              - const uint8_t *ct: pointer to input cipher text
*                (an already allocated array of KYBER_CIPHERTEXTBYTES bytes)
*              - const uint8_t *sk: pointer to input private key
*                (an already allocated array of KYBER_SECRETKEYBYTES bytes)
*
* Returns 0.
*
* On failure, ss will contain a pseudo-random value.
**************************************************/
__global__ void 
crypto_kem_dec_kernel(uint8_t *ss,
                      const uint8_t *ct,
                      const uint8_t *sk,
                      uint32_t n_keypair)
{
  const int tid = threadIdx.x;
  const int block_size = blockDim.x;
  const int bid = blockIdx.x;
  const int ss_offset = ( bid * block_size + tid ) * KYBER_SSBYTES;
  const int ct_offset = ( bid * block_size + tid ) * KYBER_CIPHERTEXTBYTES;
  const int sk_offset = ( bid * block_size + tid ) * KYBER_SECRETKEYBYTES;

  if ( bid * block_size + tid < n_keypair )
  {
      int fail;
      uint8_t buf[2*KYBER_SYMBYTES];
      /* Will contain key, coins */
      uint8_t kr[2*KYBER_SYMBYTES];
      uint8_t cmp[KYBER_CIPHERTEXTBYTES+KYBER_SYMBYTES];
      const uint8_t *pk = sk + sk_offset + KYBER_INDCPA_SECRETKEYBYTES;

      indcpa_dec(buf, ct + ct_offset, sk + sk_offset);

      /* Multitarget countermeasure for coins + contributory KEM */
      memcpy(buf+KYBER_SYMBYTES, sk + sk_offset + KYBER_SECRETKEYBYTES - 2*KYBER_SYMBYTES, KYBER_SYMBYTES);
      hash_g(kr, buf, 2*KYBER_SYMBYTES);

      /* coins are in kr+KYBER_SYMBYTES */
      indcpa_enc(cmp, buf, pk, kr+KYBER_SYMBYTES);

      fail = verify(ct + ct_offset, cmp, KYBER_CIPHERTEXTBYTES);

      /* Compute rejection key */
      rkprf(ss + ss_offset, sk + sk_offset + KYBER_SECRETKEYBYTES - KYBER_SYMBYTES, ct + ct_offset);

      /* Copy true key to return buffer if fail is false */
      cmov(ss + ss_offset, kr, KYBER_SYMBYTES, !fail);
  }
}

int crypto_kem_dec(uint8_t *ss,
                   const uint8_t *ct,
                   const uint8_t *sk,
                   uint32_t n_keypair)
{
  const uint32_t n_streams = (n_keypair + kKeysPerGrid - 1) / kKeysPerGrid;
  const uint32_t tail_keypairs = n_keypair % kKeysPerGrid;

  uint8_t *d_ss = nullptr;
  uint8_t *d_ct = nullptr;
  uint8_t *d_sk = nullptr;

  hipMalloc( &d_ss, n_keypair * KYBER_SSBYTES);
  hipMalloc( &d_ct, n_keypair * KYBER_CIPHERTEXTBYTES);
  hipMalloc( &d_sk, n_keypair * KYBER_SECRETKEYBYTES);
  assert( d_ss && d_ct && d_sk);

  std::vector<hipStream_t> streams( n_streams);
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    GPU_ASSERT( hipStreamCreate( &streams[i]));
  }

  auto getNKeysForGrid = [=]( uint32_t i) -> uint32_t
  {
    if ( (tail_keypairs != 0) && (i == n_streams - 1) )
    {
      return tail_keypairs;
    } else {
      return kKeysPerGrid;
    }
  };

  // Launch copy for arguments of every stream.
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    uint32_t offset = i * kKeysPerGrid;
    uint32_t n_cur = getNKeysForGrid( i);

    const uint8_t *h_grid_ct = ct   + offset * KYBER_CIPHERTEXTBYTES;
    const uint8_t *h_grid_sk = sk   + offset * KYBER_SECRETKEYBYTES;
    uint8_t       *d_grid_ct = d_ct + offset * KYBER_CIPHERTEXTBYTES;
    uint8_t       *d_grid_sk = d_sk + offset * KYBER_SECRETKEYBYTES;

    GPU_ASSERT( hipMemcpyAsync( d_grid_ct,
                                 h_grid_ct,
                                 n_cur * KYBER_CIPHERTEXTBYTES,
                                 hipMemcpyHostToDevice,
                                 streams[i]));

    GPU_ASSERT( hipMemcpyAsync( d_grid_sk,
                                 h_grid_sk,
                                 n_cur * KYBER_SECRETKEYBYTES,
                                 hipMemcpyHostToDevice,
                                 streams[i]));
  }

  // Launch all grids in streams.
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    uint32_t offset = i * kKeysPerGrid;
    uint32_t n_cur = getNKeysForGrid( i);

    uint8_t *d_grid_ss = d_ss + offset * KYBER_SSBYTES;
    uint8_t *d_grid_ct = d_ct + offset * KYBER_CIPHERTEXTBYTES;
    uint8_t *d_grid_sk = d_sk + offset * KYBER_SECRETKEYBYTES;

    printf( "dec gridDim (%u), blockDim (%u), stream launch (%u), #keys on grid (%u keys), offset (%u keys)\n",
            kBlocksPerGrid, kThreadsPerBlock, i, n_cur, offset);
    crypto_kem_dec_kernel<<<kBlocksPerGrid, kThreadsPerBlock, 0, streams[i]>>>(
        d_grid_ss,
        d_grid_ct,
        d_grid_sk,
        n_cur
    );
  }

  // Launch copy for result of every stream.
  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    uint32_t offset = i * kKeysPerGrid;
    uint32_t n_cur = getNKeysForGrid( i);

    uint8_t *h_grid_ss = ss   + offset * KYBER_SSBYTES;
    uint8_t *d_grid_ss = d_ss + offset * KYBER_SSBYTES;

    GPU_ASSERT( hipMemcpyAsync( h_grid_ss,
                                 d_grid_ss,
                                 n_cur * KYBER_SSBYTES,
                                 hipMemcpyDeviceToHost,
                                 streams[i]));
  }

  // Wait for all streams.
  GPU_ASSERT( hipDeviceSynchronize());

  for ( uint32_t i = 0; i < n_streams; i++ )
  {
    GPU_ASSERT( hipStreamDestroy( streams[i]));
  }

  hipFree( d_ss);
  hipFree( d_ct);
  hipFree( d_sk);

  return 0;
}
